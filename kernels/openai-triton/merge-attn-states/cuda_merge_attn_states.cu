#include "hip/hip_runtime.h"
#include <optional>
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/extension.h>
#include <algorithm>


static __forceinline__ __device__ float 
to_float(float u) { return u; }
static __forceinline__ __device__ float 
to_float(half u) { return __half2float(u); }
static __forceinline__ __device__ float 
to_float(__hip_bfloat16 u) { return __bfloat162float(u); }
static __forceinline__ __device__ void 
from_float(float& d, float s) { d = s; }
static __forceinline__ __device__ void 
from_float(half& d, float s) { d = __float2half(s); }
static __forceinline__ __device__ void 
from_float(__hip_bfloat16& d, float s) { d = __float2bfloat16(s); }


template <typename scalar_t>
__device__ __forceinline__ void merge_attn_states_per_thread(
    scalar_t* output,   // [NUM_TOKENS, NUM_HEADS, HEAD_SIZE]
    float* output_lse,  // [NUM_HEADS, NUM_TOKENS]
    const scalar_t* __restrict__ prefix_output,  // [NUM_TOKENS, NUM_HEADS,
                                                 // HEAD_SIZE]
    const float* __restrict__ prefix_lse,        // [NUM_HEADS, NUM_TOKENS]
    const scalar_t* __restrict__ suffix_output,  // [NUM_TOKENS, NUM_HEADS,
                                                 // HEAD_SIZE]
    const float* __restrict__ suffix_lse,        // [NUM_HEADS, NUM_TOKENS]
    const uint num_tokens,                       // NUM_TOKENS
    const uint num_heads,                        // NUM QUERY HEADS
    const uint head_size,  // HEAD_SIZE, 32,48,64,...,512,etc
    const uint token_idx,
    const uint head_idx,
    const uint thr_idx
) {
  using pack_128b_t = uint4; // float -> 4, half/bf16 -> 8
  constexpr uint pack_size = 16 / sizeof(scalar_t);

  const uint thr_offset = thr_idx * pack_size; // (0~15)*8, etc.
  const uint blk_offset =
    token_idx * num_heads * head_size + head_idx * head_size;
  const scalar_t* prefix_output_blk = prefix_output + blk_offset;
  const scalar_t* suffix_output_blk = suffix_output + blk_offset;
  scalar_t* output_blk = output + blk_offset;

  float p_lse = prefix_lse[head_idx * num_tokens + token_idx];
  float s_lse = suffix_lse[head_idx * num_tokens + token_idx];
  p_lse = std::isinf(p_lse) ? -std::numeric_limits<float>::infinity() : p_lse;
  s_lse = std::isinf(s_lse) ? -std::numeric_limits<float>::infinity() : s_lse;

  const float max_lse = fmaxf(p_lse, s_lse);
  p_lse = p_lse - max_lse;
  s_lse = s_lse - max_lse;
  const float p_se = expf(p_lse);
  const float s_se = expf(s_lse);
  const float out_se = p_se + s_se;
  const float p_scale = p_se / out_se;
  const float s_scale = s_se / out_se;

  // We only need to write to output_lse once per head.
  if (output_lse != nullptr && thr_idx == 0) {
    float out_lse = logf(out_se) + max_lse;
    output_lse[head_idx * num_tokens + token_idx] = out_lse;
  }

  if (thr_offset < head_size) {
    // Pack 128b load
    pack_128b_t p_out_pack = reinterpret_cast<const pack_128b_t*>(
        prefix_output_blk)[thr_offset / pack_size];
    pack_128b_t s_out_pack = reinterpret_cast<const pack_128b_t*>(
        suffix_output_blk)[thr_offset / pack_size];
    pack_128b_t o_out_pack;

    #pragma unroll
    for (uint i = 0; i < pack_size; ++i) {
      // Always use float for FMA to keep precision.
      // half(uint16_t), bfloat16, float -> float.
      const float p_out_f =
          to_float(reinterpret_cast<const scalar_t*>(&p_out_pack)[i]);
      const float s_out_f =
          to_float(reinterpret_cast<const scalar_t*>(&s_out_pack)[i]);
      // fma: a * b + c = p_out_f * p_scale + (s_out_f * s_scale)
      const float o_out_f = p_out_f * p_scale + (s_out_f * s_scale);
      // float -> half(uint16_t), bfloat16, float.
      from_float(reinterpret_cast<scalar_t*>(&o_out_pack)[i],
                 o_out_f);
    }

    // Pack 128b storage
    reinterpret_cast<pack_128b_t*>(output_blk)[
      thr_offset / pack_size] = o_out_pack;
  }
}

// Implements section 2.2 of https://www.arxiv.org/pdf/2501.01005
// can be used to combine partial attention results (in the split-KV case)
template <typename scalar_t, bool kLoopOverHead, bool kFlattenOverHead = false>
__global__ void merge_attn_states_kernel(
    scalar_t* output,   // [NUM_TOKENS, NUM_HEADS, HEAD_SIZE]
    float* output_lse,  // [NUM_HEADS, NUM_TOKENS]
    const scalar_t* __restrict__ prefix_output,  // [NUM_TOKENS, NUM_HEADS,
                                                 // HEAD_SIZE]
    const float* __restrict__ prefix_lse,        // [NUM_HEADS, NUM_TOKENS]
    const scalar_t* __restrict__ suffix_output,  // [NUM_TOKENS, NUM_HEADS,
                                                 // HEAD_SIZE]
    const float* __restrict__ suffix_lse,        // [NUM_HEADS, NUM_TOKENS]
    const uint num_tokens,                       // NUM_TOKENS
    const uint num_heads,                        // NUM QUERY HEADS
    const uint head_size  // HEAD_SIZE, 32,48,64,...,512,etc
) {
  if constexpr (kLoopOverHead) {
    // May loop over num heads for large num_tokens
    const uint token_idx = blockIdx.x;
    const uint thread_idx = threadIdx.x;
    
    if constexpr (kFlattenOverHead) {
      // thread num = (num_heads * head_size) / pack_size 
      // = num_heads * (head_size / pack_size), 16 * (128 / 8)
      // tid: 0~255, 0~15->head 0, 16~31->head 1, ..., etc.
      constexpr uint pack_size = 16 / sizeof(scalar_t);
      const uint head_idx = thread_idx / (head_size / pack_size);
      const uint thr_idx = thread_idx % (head_size / pack_size);
      merge_attn_states_per_thread<scalar_t>(
        output, output_lse, prefix_output, 
        prefix_lse, suffix_output, suffix_lse, 
        num_tokens, num_heads, head_size,
        token_idx, head_idx, thr_idx
      );
    } else {
      const uint thr_idx = thread_idx;
      #pragma unroll
      for (uint head_idx = 0; head_idx < num_heads; ++head_idx) {
        merge_attn_states_per_thread<scalar_t>(
          output, output_lse, prefix_output, 
          prefix_lse, suffix_output, suffix_lse, 
          num_tokens, num_heads, head_size,
          token_idx, head_idx, thr_idx
        );
      }  // End loop over heads
    } // End kFlattenOverHead

  } else {
    const uint token_idx = blockIdx.x;
    const uint head_idx = blockIdx.y;
    const uint thread_idx = threadIdx.x;
    const uint thr_idx = thread_idx;

    merge_attn_states_per_thread<scalar_t>(
      output, output_lse, prefix_output, 
      prefix_lse, suffix_output, suffix_lse, 
      num_tokens, num_heads, head_size,
      token_idx, head_idx, thr_idx
    );
  }
}

// The following macro is used to dispatch the conversion function based on
// the output data type. The FN is a macro that calls a function with
// template<typename SCALAR_T>.
#define DISPATCH_BY_SCALAR_DTYPE(SCALAR_DTYPE, FN)                            \
  {                                                                           \
    if (SCALAR_DTYPE == at::ScalarType::Float) { FN(float); }                 \
    else if (SCALAR_DTYPE == at::ScalarType::Half) { FN(half); }              \
    else if (SCALAR_DTYPE == at::ScalarType::BFloat16) { FN(__hip_bfloat16); } \
    else {                                                                    \
      TORCH_CHECK(false, "Unsupported data type of O: ", SCALAR_DTYPE);       \
    }                                                                         \
  }

#define LAUNCH_MERGE_ATTN_STATES(SCALAR_T, kLoopOverHead, kFlattenOverHead) \
  {                                                                         \
    merge_attn_states_kernel<SCALAR_T, kLoopOverHead, kFlattenOverHead>     \
        <<<grid, block>>>(                                                  \
            reinterpret_cast<SCALAR_T*>(output.data_ptr()), output_lse_ptr, \
            reinterpret_cast<SCALAR_T*>(prefix_output.data_ptr()),          \
            reinterpret_cast<float*>(prefix_lse.data_ptr()),                \
            reinterpret_cast<SCALAR_T*>(suffix_output.data_ptr()),          \
            reinterpret_cast<float*>(suffix_lse.data_ptr()), num_tokens,    \
            num_heads, head_size);                                          \
  }

template <typename SCALAR_T>
void merge_attn_states_launcher(
    torch::Tensor& output,  // [NUM_TOKENS, NUM_HEADS, HEAD_SIZE]
    std::optional<torch::Tensor> output_lse,  // [NUM_HEADS, NUM_TOKENS]
    const torch::Tensor& prefix_output,  // [NUM_TOKENS, NUM_HEADS, HEAD_SIZE]
    const torch::Tensor& prefix_lse,     // [NUM_HEADS, NUM_TOKENS]
    const torch::Tensor& suffix_output,  // [NUM_TOKENS, NUM_HEADS, HEAD_SIZE]
    const torch::Tensor& suffix_lse,     // [NUM_HEADS, NUM_TOKENS]
    const bool disable_loop_over_head) {
  const uint num_tokens = output.size(0);
  const uint num_heads = output.size(1);  // num query heads
  const uint head_size = output.size(2);
  // float -> 4, half/bf16 -> 8, 128b = 16 bytes.
  constexpr uint pack_size = 16 / sizeof(SCALAR_T);
  TORCH_CHECK(head_size % pack_size == 0,
              "headsize must be multiple of pack_size:", pack_size);
  TORCH_CHECK(head_size / pack_size <= 1024,
              "headsize/pack_size must be <= of 1024, pack_size: ", pack_size);
  float* output_lse_ptr = nullptr;
  if (output_lse.has_value()) {
    output_lse_ptr = output_lse.value().data_ptr<float>();
  }
  // Keep threads num <= 512 per thread block.
  const bool skip_flatten_over_head = (
    (num_heads * head_size) / pack_size > 512);

  const bool skip_loop_over_head = (
    disable_loop_over_head || num_tokens <= 1024 || 
    (num_heads >= 64 && skip_flatten_over_head)
  );

  if (skip_loop_over_head) {
    dim3 grid(num_tokens, num_heads);
    dim3 block(head_size / pack_size);
    LAUNCH_MERGE_ATTN_STATES(SCALAR_T, false, false);
  } else {
    // try loop over num heads for large num_tokens
    if (skip_flatten_over_head) {
      dim3 grid(num_tokens);
      dim3 block(head_size / pack_size);
      LAUNCH_MERGE_ATTN_STATES(SCALAR_T, true, false);
    } else {
      // cases:
      // num_tokens 8192, num_heads 16, head_size 128
      // num_tokens 4096, num_heads 16, head_size 128
      dim3 grid(num_tokens);
      dim3 block((num_heads * head_size) / pack_size);
      LAUNCH_MERGE_ATTN_STATES(SCALAR_T, true, true);
    }
  }
}

#define CALL_MERGE_ATTN_STATES_LAUNCHER(SCALAR_T) \
  {                                               \
    merge_attn_states_launcher<SCALAR_T>(         \
        output, output_lse, prefix_output,        \
        prefix_lse, suffix_output,                \
        suffix_lse, disable_loop_over_head);      \
  }

void merge_attn_states_cuda(
    torch::Tensor& output,  // [NUM_TOKENS, NUM_HEADS, HEAD_SIZE]
    std::optional<torch::Tensor> output_lse,  // [NUM_HEADS, NUM_TOKENS]
    const torch::Tensor& prefix_output,  // [NUM_TOKENS, NUM_HEADS, HEAD_SIZE]
    const torch::Tensor& prefix_lse,     // [NUM_HEADS, NUM_TOKENS]
    const torch::Tensor& suffix_output,  // [NUM_TOKENS, NUM_HEADS, HEAD_SIZE]
    const torch::Tensor& suffix_lse,     // [NUM_HEADS, NUM_TOKENS]
    const bool disable_loop_over_head) {
  DISPATCH_BY_SCALAR_DTYPE(output.dtype(), CALL_MERGE_ATTN_STATES_LAUNCHER);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def(
    "merge_attn_states_cuda", 
    &merge_attn_states_cuda, 
    py::arg("output"),
    py::arg("output_lse").none(true),
    py::arg("prefix_output"),
    py::arg("prefix_lse"),
    py::arg("suffix_output"),
    py::arg("suffix_lse"),
    py::arg("disable_loop_over_head"),
    "Merge attention states (CUDA)"
  );
}